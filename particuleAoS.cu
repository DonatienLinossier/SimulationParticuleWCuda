#include "hip/hip_runtime.h"
#include "particuleAoS.cuh"


particuleAoS::particuleAoS()
{
	nbThread = 0;
	nbBlock = 0;
	width = 0;
	height = 0;
	GRIDSIZE = 0;
	CASEMAXX = 0;
	CASEMAXY = 0;
	SIZECASEX = 0;
	SIZECASEY = 0;
	nbParticule = 0;
	dev_x = nullptr;
	dev_y = nullptr;
	dev_lastx = nullptr;
	dev_lasty = nullptr;
	dev_vx = nullptr;
	dev_vy = nullptr;
	dev_cell = nullptr;
	dev_color = nullptr;
	dev_index = nullptr;
	dev_radius = nullptr;
	dev_changed = nullptr;
};
particuleAoS::particuleAoS(int gridSize, int wt, int ht, int nbcaseX, int nbcaseY, int cw, int ch)
{
	nbThread = NBTHREAD;
	nbBlock = 0;
	width = wt;
	height = ht;
	GRIDSIZE = gridSize;
	CASEMAXX = nbcaseX;
	CASEMAXY = nbcaseY;
	SIZECASEX = cw;
	SIZECASEY = ch;
	nbParticule = 0;
	dev_alloc = nullptr;
	dev_x = nullptr;
	dev_y = nullptr;
	dev_lastx = nullptr;
	dev_lasty = nullptr;
	dev_vx = nullptr;
	dev_vy = nullptr;
	dev_cell = nullptr;
	dev_color = nullptr;
	dev_index = nullptr;
	dev_radius = nullptr;
	dev_changed = nullptr;
};


__device__ cell particuleAoS::dev_toCell(int index, float SIZECASEX, float SIZECASEY, int CASEMAXX, int CASEMAXY) {


	if (index >= nbParticule) {
		return;
	}

	dev_tension[index] = 0;

	int cx = dev_x[index] / SIZECASEX;
	int cy = dev_y[index] / SIZECASEY;


	cell retoure;
	retoure.x = dev_cell[index].x;
	retoure.y = dev_cell[index].y;
	if (cx != dev_cell[index].x || cy != dev_cell[index].y) {
		//p_actualiazed = false;

		dev_cell[index].x = cx;
		dev_cell[index].y = cy;

		if (dev_cell[index].x < 0) {
			dev_cell[index].x = 0;
		}
		else if (dev_cell[index].x >= CASEMAXX) {
			dev_cell[index].x = CASEMAXX - 1;
		}

		if (dev_cell[index].y < 0) {
			dev_cell[index].y = 0;
		}
		else if (dev_cell[index].y >= CASEMAXY) {
			dev_cell[index].y = CASEMAXY - 1;
		}
	}
	return retoure;



}

void particuleAoS::forceOnPoint(int x, int y, float dt, int intensite) {
	forceOnPoint_global << <nbBlock, nbThread >> > (dev_x, dev_y, dev_vx, dev_vy, nbParticule, dt, x, y, intensite);
}

void particuleAoS::force(float dt) {
	force_global << <nbBlock, nbThread >> > (dev_x, dev_y, dev_lastx, dev_lasty, dev_vx, dev_vy, nbParticule, dt);
}

void particuleAoS::CalcPosition(float dt) {
	CalcPosition_global << <nbBlock, nbThread >> > (dev_x, dev_y, dev_lastx, dev_lasty, dev_vx, dev_vy, nbParticule, dt);
}

void particuleAoS::toCell(float SIZECASEX, float SIZECASEY, int CASEMAXX, int CASEMAXY) {
	toCell_global<<<nbBlock, nbThread>>>(dev_x, dev_y, dev_cell, dev_lastCell, nbParticule, SIZECASEX, SIZECASEY, CASEMAXX, CASEMAXY);
}

void particuleAoS::borderCollision() {
	borderCollision_global << <nbBlock, nbThread >> > (dev_x, dev_y, dev_radius, width, height, nbParticule);
}

__global__ void displayData(float* data, float* data2, float* data3, float* data4, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index > size) {
		return;
	}
	printf("%lf %lf %lf %lf\n", data[index], data2[index], data3[index], data4[index]);
}

void particuleAoS::addParticules(int nbNewParticules) {


	void* cpy_alloc = dev_alloc;

	size_t floatSize = (nbParticule + nbNewParticules) * sizeof(float);
	size_t intSize = (nbParticule + nbNewParticules) * sizeof(int);
	size_t boolSize = (nbParticule + nbNewParticules) * sizeof(bool);
	size_t cellSize = (nbParticule + nbNewParticules) * sizeof(cell);
	size_t uchar4_tSize = (nbParticule + nbNewParticules) * sizeof(uchar4);

	hipMalloc((void**)&dev_alloc,
		floatSize * 7 +  // 6 arrays of floats
		intSize * 2 +  // 3 arrays of ints
		cellSize * 2 +
		uchar4_tSize * 1 +
		boolSize);       // 1 array of bools

	// Assign pointers to different parts of the allocated memory
	dev_x = reinterpret_cast<float*>(dev_alloc);
	dev_y = dev_x + (nbParticule + nbNewParticules);
	dev_lastx = dev_y + (nbParticule + nbNewParticules);
	dev_lasty = dev_lastx + (nbParticule + nbNewParticules);
	dev_vx = dev_lasty + (nbParticule + nbNewParticules);
	dev_vy = dev_vx + (nbParticule + nbNewParticules);
	dev_tension = dev_vy + (nbParticule + nbNewParticules);
	dev_cell = reinterpret_cast<cell*>(dev_tension + (nbParticule + nbNewParticules));
	dev_lastCell = reinterpret_cast<cell*>(dev_lastCell + (nbParticule + nbNewParticules));
	dev_color = reinterpret_cast<uchar4*>(dev_cell + (nbParticule + nbNewParticules));
	dev_index = reinterpret_cast<int*>(dev_color + (nbParticule + nbNewParticules));
	dev_radius = dev_index + (nbParticule + nbNewParticules);
	dev_changed = reinterpret_cast<bool*>(dev_radius + (nbParticule + nbNewParticules));

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("\n---------AddParticules before lalala - : %s\n", hipGetErrorString(cudaStatus));
	}

	//Copie des anciens elements
	if (cpy_alloc != nullptr) {
		void* offset = cpy_alloc;
		hipMemcpy(dev_x, offset, nbParticule * sizeof(float), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<float*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_y, offset, nbParticule * sizeof(float), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<float*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_lastx, offset, nbParticule * sizeof(float), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<float*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_lasty, offset, nbParticule * sizeof(float), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<float*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_vx, offset, nbParticule * sizeof(float), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<float*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_vy, offset, nbParticule * sizeof(float), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<float*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_tension, offset, nbParticule * sizeof(float), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<float*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_cell, offset, nbParticule * sizeof(cell), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<cell*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_lastCell, offset, nbParticule * sizeof(cell), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<cell*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_color, offset, nbParticule * sizeof(uchar4), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<uchar4*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_index, offset, nbParticule * sizeof(int), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<int*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_radius, offset, nbParticule * sizeof(int), hipMemcpyDeviceToDevice);

		offset = reinterpret_cast<int*>(offset) + (nbParticule + nbNewParticules);
		hipMemcpy(dev_changed, offset, nbParticule * sizeof(bool), hipMemcpyDeviceToDevice);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("\n---------AddParticules in lalala - : %s\n", hipGetErrorString(cudaStatus));
		}

		hipFree(cpy_alloc);
	}



	//Creation des nouveaux elements
	float* new_x = (float*)malloc(nbNewParticules * sizeof(float));
	float* new_y = (float*)malloc(nbNewParticules * sizeof(float));
	float* new_lastx = (float*)malloc(nbNewParticules * sizeof(float));
	float* new_lasty = (float*)malloc(nbNewParticules * sizeof(float));
	float* new_vx = (float*)malloc(nbNewParticules * sizeof(float));
	float* new_vy = (float*)malloc(nbNewParticules * sizeof(float));
	cell* new_cell = (cell*)malloc(nbNewParticules * sizeof(cell));
	cell* new_lastCell = (cell*)malloc(nbNewParticules * sizeof(cell));
	uchar4* new_color = (uchar4*)malloc(nbNewParticules * sizeof(uchar4));
	int* new_index = (int*)malloc(nbNewParticules * sizeof(int));
	int* new_radius = (int*)malloc(nbNewParticules * sizeof(int));
	float* new_tension = (float*)malloc(nbNewParticules * sizeof(float));
	bool* new_bool = (bool*)malloc(nbNewParticules * sizeof(bool));
	for (int i = 0; i < nbNewParticules; ++i) {
		new_x[i] = (float) 5 + std::rand() % (width - 10);
		new_y[i] = (float) 5 + std::rand() % (height - 10);
		new_vx[i] = 0;
		new_vy[i] = 0;
		new_lastx[i] = new_x[i] - new_vx[i];
		new_lasty[i] = new_y[i] - new_vy[i];
		new_cell[i] = { -1, -1 };
		new_lastCell[i] = { -1, -1 }; 
		new_color[i] = { static_cast<unsigned char>((new_y[i] / height) * 255) , static_cast<unsigned char>(255-(new_y[i] / height) * 255), static_cast<unsigned char>(255 - (new_x[i] / width) * 255), 0};
		new_index[i] = -1;
		new_radius[i] = PARTICULE_SIZE;
		new_tension[i] = 0;
		new_bool[i] = false;
	}



	//Transfert des nouveaux elements sur le GPU
	hipMemcpy(dev_x + nbParticule, new_x, nbNewParticules * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_y + nbParticule, new_y, nbNewParticules * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_lastx + nbParticule, new_lastx, nbNewParticules * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_lasty + nbParticule, new_lasty, nbNewParticules * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vx + nbParticule, new_vx, nbNewParticules * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vy + nbParticule, new_vy, nbNewParticules * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_tension + nbParticule, new_tension, nbNewParticules * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_cell + nbParticule, new_cell, nbNewParticules * sizeof(cell), hipMemcpyHostToDevice);
	hipMemcpy(dev_lastCell + nbParticule, new_lastCell, nbNewParticules * sizeof(cell), hipMemcpyHostToDevice);
	hipMemcpy(dev_color + nbParticule, new_color, nbNewParticules * sizeof(uchar4), hipMemcpyHostToDevice);
	hipMemcpy(dev_index + nbParticule, new_index, nbNewParticules * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_radius + nbParticule, new_radius, nbNewParticules * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_changed + nbParticule, new_bool, nbNewParticules * sizeof(bool), hipMemcpyHostToDevice);
	//displayData << <1, 50 >> > (dev_x, dev_y, dev_vx, dev_vy, 50);
	free(new_x);
	free(new_y);
	free(new_lastx);
	free(new_lasty);
	free(new_vx);
	free(new_vy);
	free(new_cell);
	free(new_lastCell);
	free(new_color);
	free(new_index);
	free(new_radius);
	free(new_tension);
	free(new_bool);

	nbParticule += nbNewParticules;

	nbBlock = (nbParticule + nbThread - 1) / nbThread;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("\n---------AddParticules - : %s\n", hipGetErrorString(cudaStatus));
	} 

}



__global__ void global_drawDotsBis(uint32_t* buf, int width, int height, float* tab_x, float* tab_y, int* tab_color, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index > size) {
		return;
	}


	int x = (int) tab_x[index];
	int y = (int) tab_y[index];
	int color = tab_color[index];

	int pos = y * width + x;
	if (y >= height || y < 0 || x >= width || x < 0) {
		return;
	}

	buf[pos] = (uint32_t)color;
}

void particuleAoS::GPUdraw_point(uint32_t* buf, int width, int height) {
	//drawDots(system_, (int*) dev_x, (int*) dev_y, (int*) dev_color, nbParticule);
	int size = nbParticule;

	int nbthread = 1024;
	int numBlocks = (size + nbthread - 1) / nbthread;

	global_drawDotsBis << <numBlocks, nbthread >> > (buf, width, height, dev_x, dev_y, (int*)dev_color, size);
}

__global__ void global_drawDotsBisNew(uchar4* dev_gpuPixels, int width, int height, float* tab_x, float* tab_y, int* tab_color, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size) {
		return;
	}

	float x_d = tab_x[index];
	float y_d = tab_y[index];
	int color = tab_color[index];

	int x = static_cast<int>(x_d);
	int y = static_cast<int>(y_d);

	if (y >= height || y < 0 || x >= width || x < 0) {
		return;
	}

	//size_t pos = y * width + x;
	uchar4* element = dev_gpuPixels + y * width + x;
	element->x = 255; // Red channel
	element->y = 0;   // Green channel
	element->z = 0;   // Blue channel
	element->w = 255; // Alpha channel
}

void particuleAoS::GPUdraw_pointNew(uchar4* dev_gpuPixels, int width, int height) {
	//drawDots(system_, (int*) dev_x, (int*) dev_y, (int*) dev_color, nbParticule);
	int size = nbParticule;

	int nbthread = 1024;
	int numBlocks = (size + nbthread - 1) / nbthread;

	global_drawDotsBisNew << <numBlocks, nbthread >> > (dev_gpuPixels, width, height, dev_x, dev_y, (int*)dev_color, size);

	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();

	if (cudaStatus != hipSuccess) {
		printf("Cuda error -> heuuu nouveau:%s\n", hipGetErrorString(cudaStatus));
	}
}


__device__ void drawCircle_arcs(int xc, int yc, int x, int y, uchar4* dev_gpuPixels, int width, int height, uchar4 color)
{
	//printf("color %d", color);
	/*
	each block is an octant

	*/
	int yc_add_y = yc + y;
	int yc_sub_y = yc - y;
	int xc_add_x = xc + x;
	int xc_sub_x = xc - x;

	int yc_add_x = yc + x;
	int yc_sub_x = yc - x;
	int xc_add_y = xc + y;
	int xc_sub_y = xc - y;

	uchar4* limite = dev_gpuPixels + height * width;


	if (!(yc_add_y >= height)) {

		//int pos = yc_add_y * width + xc_add_x;

		if (!(xc_add_x >= width)) {
			uchar4* element = dev_gpuPixels + yc_add_y * width + xc_add_x;
			if (element >= limite)
				return;
			*element = color;
		}

		//pos = yc_add_y * width + xc_sub_x;
		if (!(xc_sub_x < 0)) {
			uchar4* element = dev_gpuPixels + yc_add_y * width + xc_sub_x;
			if (element >= limite)
				return;
			*element = color;
		}

	}

	if (!(yc_sub_y < 0)) {
		//int pos = yc_sub_y * width + xc_add_x;
		if (!(xc_add_x >= width)) {
			uchar4* element = dev_gpuPixels + yc_sub_y * width + xc_add_x;
			if (element >= limite)
				return;
			*element = color;
		}

		//pos = yc_sub_y * width + xc_sub_x;
		if (!(xc_sub_x < 0)) {
			uchar4* element = dev_gpuPixels + yc_sub_y * width + xc_sub_x;
			if (element >= limite)
				return;
			*element = color;
		}
	}




	if (!(yc_add_x >= height)) {
		//int pos = yc_add_x * width + xc_add_y;
		if (!(xc_add_y >= width)) {
			uchar4* element = dev_gpuPixels + yc_add_x * width + xc_add_y;
			if (element >= limite)
				return;
			*element = color;
		}

		//pos = yc_add_x * width + xc_sub_y;
		if (!(xc_sub_y < 0)) {
			uchar4* element = dev_gpuPixels + yc_add_x * width + xc_sub_y;
			if (element >= limite)
				return;
			*element = color;
		}

	}

	if (!(yc_sub_x < 0)) {
		//int pos = yc_sub_x * width + xc_add_y;
		if (!(xc_add_y >= width)) {
			uchar4* element = dev_gpuPixels + yc_sub_x * width + xc_add_y;
			if (element >= limite)
				return;
			*element = color;
		}

		//pos = yc_sub_x * width + xc_sub_y;
		if (!(xc_sub_y < 0)) {
			uchar4* element = dev_gpuPixels + yc_sub_x * width + xc_sub_y;
			if (element >= limite)
				return;
			*element = color;
		}
	}
}

__global__ void global_drawCircleNew(uchar4* dev_gpuPixels, int width, int height, float* tab_x, float* tab_y, int* tab_radius, uchar4* tab_color, float* dev_tension, int size) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size) {
		return;
	}

	int xc = tab_x[index];
	int yc = tab_y[index];
	int r = tab_radius[index];
	//int color = tab_color[index];
	uchar4 color = tab_color[index];// (int)(dev_tension[index] * 255) % 255;

	int x = 0, y = r;
	int d = 3 - 2 * r;
	drawCircle_arcs(xc, yc, x, y, dev_gpuPixels, width, height, color);
	while (y >= x)
	{
		// for each pixel we will
		// draw all eight pixels

		x++;

		// check for decision parameter
		// and correspondingly
		// update d, x, y
		if (d > 0)
		{
			y--;
			d = d + 4 * (x - y) + 10;
		}
		else
			d = d + 4 * x + 6;
		drawCircle_arcs(xc, yc, x, y, dev_gpuPixels, width, height, color);
	}

}

void particuleAoS::GPUdraw_CircleNew(uchar4* dev_gpuPixels, int width, int height) {
	//drawDots(system_, (int*) dev_x, (int*) dev_y, (int*) dev_color, nbParticule);
	int size = nbParticule;

	int nbthread = 1024;
	int numBlocks = (size + nbthread - 1) / nbthread;

	global_drawCircleNew << <numBlocks, nbthread >> > (dev_gpuPixels, width, height, dev_x, dev_y, dev_radius, dev_color, dev_tension, size);

	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();

	if (cudaStatus != hipSuccess) {
		printf("Cuda error -> heuuu nouveau:%s\n", hipGetErrorString(cudaStatus));
	}
}

__global__ void global_GPUdrawFilledCircle(uchar4* dev_gpuPixels, int width, int height, float* dev_x, float* dev_y, int* dev_radius, uchar4* dev_color, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size) {
		return;
	}

	int xc = dev_x[index];
	int yc = dev_y[index];
	int radius = dev_radius[index];
	uchar4 color = dev_color[index];
	int rSquared = radius * radius;

	uchar4* limite = dev_gpuPixels + height * width;

	for (int y = -radius; y <= radius; y++) {
		for (int x = -radius; x <= radius; x++) {
			if (x * x + y * y <= rSquared) {
				int xpos = xc + x;
				int ypos = yc + y;

				if (xpos >= 0 && xpos < width && ypos >= 0 && ypos < height) {
					uchar4* element = dev_gpuPixels + ypos * width + xpos;
					if (element >= limite)
						continue;
					*element = color;
				}
			}
		}
	}
}

void particuleAoS::GPUdrawFilledCircle(uchar4* dev_gpuPixels, int width, int height) {
	int size = nbParticule;

	int nbthread = 1024;
	int numBlocks = (size + nbthread - 1) / nbthread;

	global_GPUdrawFilledCircle << <numBlocks, nbthread >> > (dev_gpuPixels, width, height, dev_x, dev_y, dev_radius, dev_color, size);

	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();

	if (cudaStatus != hipSuccess) {
		printf("Cuda error -> heuuu nouveau:%s\n", hipGetErrorString(cudaStatus));
	}
}





__device__ cell particuleAoS::getCell(int index) {
	if (index >= nbParticule) {
		printf("\nmaaahaha");
		return { -1, -1 };
	}
	return dev_cell[index];
}

__device__ void particuleAoS::setChanged(int index, bool newValue) {
	if (index >= nbParticule) {
		return;
	}
	dev_changed[index] = newValue;
}

__device__ bool particuleAoS::getChanged(int index) {
	if (index >= nbParticule) {
		return false;
	}
	return dev_changed[index];
}

__device__ void particuleAoS::setIndex(int index, int newValue) {
	if (index >= nbParticule) {
		return;
	}
	dev_index[index] = newValue;
}

__device__ int particuleAoS::getIndex(int index) {
	if (index >= nbParticule) {
		return -1;
	}
	return dev_index[index];
}

__device__ void particuleAoS::setRadius(int index, float newValue) {
	if (index >= nbParticule) {
		return;
	}
	dev_radius[index] = newValue;
}

__device__ int particuleAoS::getRadius(int index) {
	if (index >= nbParticule) {
		return -1; // Return a default value or an error indicator if out of bounds.
	}
	return dev_radius[index];
}

__device__ void particuleAoS::setTension(int index, float newValue) {
	if (index >= nbParticule)
		return; 

	dev_tension[index] = newValue;
}

__device__ float particuleAoS::getTension(int index) {
	if (index >= nbParticule) {
		return -1.0f; // Return a default value or an error indicator if out of bounds.
	}
	return dev_tension[index];
}

__device__ void particuleAoS::setX(int index, float newValue) {
	if (index >= nbParticule) {
		return;
	}
	dev_x[index] = newValue;
}

__device__ float particuleAoS::getX(int index) {
	if (index >= nbParticule) {
		return -1.0; // Return a default value or an error indicator if out of bounds.
	}
	return dev_x[index];
}

__device__ void particuleAoS::setY(int index, float newValue) {
	if (index >= nbParticule) {
		return;
	}

	//printf("\n %d %f", index, newValue);
	dev_y[index] = newValue;
}

__device__ float particuleAoS::getY(int index) {
	if (index >= nbParticule) {
		return -1.0; // Return a default value or an error indicator if out of bounds.
	}
	return dev_y[index];
}